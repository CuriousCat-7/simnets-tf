#include "hip/hip_runtime.h"
//
// Created by elhanani on 01/04/17.
//

#include "mex_kernel_common.hpp"
#include "ggemm_cpu.hpp"
#include "ggemm.cuh"

using namespace tensorflow;

template <typename Dtype>
void mex_forward_gpu(const int M, const int N, const int K, const bool softmax_mode,
                     const Dtype epsilon, const Dtype* offsets, const Dtype* in, Dtype* out, const int batch_size = 1) {
    const Dtype init_value = epsilon > 0 ? -INFINITY : INFINITY;
    if (epsilon > 0) {
        ggemm_gpu
                <Dtype, Dtype, Dtype, uint8_t,
                        ggemm_add<Dtype, uint8_t>, ggemm_max<Dtype>, false,
                        true, true, true>
                (M, N, K, offsets, in, out,
                 init_value, init_value, init_value, 0, batch_size);
    } else {
        ggemm_gpu
                <Dtype, Dtype, Dtype, uint8_t,
                        ggemm_add<Dtype, uint8_t>, ggemm_min<Dtype>, false,
                        true, true, true>
                (M, N, K, offsets, in, out,
                 init_value, init_value, init_value, 0, batch_size);
    }
    if (std::isfinite(epsilon)) {
        ggemm_readc_gpu
                <false, false, Dtype, Dtype, Dtype, typename vec<Dtype>::vec2,
                        mex_forward_exp<Dtype>, ggemm_add<Dtype>, true, mex_forward_out<Dtype>, true,
                        true, true, true>
                (M, N, K, offsets, in, out, out,
                 init_value, init_value, 0, make_vec2<Dtype>(epsilon, softmax_mode ? Dtype(0) : (Dtype)-std::log(K)), batch_size);
    }
}
namespace
{
    template <typename T, typename D>
    void copy_with_eigen(T* dest, const T* source, size_t sz, const D& eigen_device)
    {
        typename TTypes<T,1>::ConstTensor src(source, sz);
        typename TTypes<T,1>::Tensor dst(dest, sz);
        dst.device(eigen_device) = src;
    }
}

template<typename T>
class MEXKernelGPU : public MEXKernelCommon {
public:

    using Base = MEXKernelCommon;
    using Dtype = T;

    MEXKernelGPU(OpKernelConstruction *context) : Base(context) {}

    void Compute(OpKernelContext *context) override {
        CalculateDimensionsWithConext(context);

        auto input = context->input(0);
        auto offsets_unpadded = context->input(1);
        auto input_t = input.tensor<T, 4>();
        auto offsets_unpadded_t = offsets_unpadded.tensor<T, 5>();

        Tensor offsets_padded;
        TensorShape offsets_padded_shape{{offsets_unpadded_t.size() + ggemm_padded_output_size(M_, K_)}};
        context->allocate_temp(DataTypeToEnum<T>::value, offsets_padded_shape, &offsets_padded);
        auto offsets_padded_t = offsets_padded.tensor<T, 1>();
        copy_with_eigen(offsets_padded_t.data(), offsets_unpadded_t.data(),
                        offsets_unpadded_t.size(), context->eigen_gpu_device());


        Tensor *output = NULL;

        TensorShape output_shape{batch_, channels_out_total_, height_out_, width_out_};
        OP_REQUIRES_OK(context, context->allocate_output(0, output_shape, &output));
        auto output_t = output->tensor<T, 4>();

        Tensor col_buffer;
        TensorShape col_buffer_shape{{K_ * channels_out_ * height_out_ * width_out_ + ggemm_padded_output_size(K_, N_)}};
        context->allocate_temp(DataTypeToEnum<T>::value, col_buffer_shape, &col_buffer);
        auto col_buffer_t = col_buffer.tensor<T, 1>();

        Tensor split_patches_in_tensor;
        TensorShape split_patches_in_shape{{num_regions_ * K_ * region_size_ + ggemm_padded_output_size(K_, region_size_)}};
        context->allocate_temp(DataTypeToEnum<T>::value, split_patches_in_shape, &split_patches_in_tensor);
        auto split_patches_in_t = split_patches_in_tensor.tensor<T, 1>();

        Tensor split_patches_out_tensor;
        TensorShape split_patches_out_shape{{num_regions_ * M_ * region_size_ + ggemm_padded_output_size(M_, region_size_)}};
        context->allocate_temp(DataTypeToEnum<T>::value, split_patches_out_shape, &split_patches_out_tensor);
        auto split_patches_out_t = split_patches_out_tensor.tensor<T, 1>();

        // -------------------------------------------------------------------------------

        const Dtype *col_buff = NULL;

        if (!is_1x1_) {
            col_buff = col_buffer_t.data();
        }
        const Dtype epsilon = epsilon_;
        const Dtype *split_patches_in = NULL;
        Dtype *split_patches_out = NULL;
        const Dtype *offsets = offsets_padded_t.data();

        auto input_at_batch = [&](int n) {
            return input_t.data() + n * channels_ * height_ * width_;
        };

        auto output_at_batch = [&](int n) {
            return output_t.data() + n * channels_out_total_ * height_out_ * width_out_;
        };

        for (int n = 0; n < batch_; ++n) {
            // im2col transformation: unroll input regions for filtering
            // into column matrix for multplication.
            if (!is_1x1_) {
                simnets_tf::im2col_3d_gpu<T>(
                        input_at_batch(n),
                        channels_, height_, width_,
                        block_c_, block_h_, block_w_,
                        pad_c_, pad_h_, pad_w_,
                        stride_c_, stride_h_, stride_w_,
                        col_buffer_t.data(),
                        blocks_round_down_, blocks_out_of_bounds_value_);
            } else {  // special case for 1x1 convolution
                col_buff = input_at_batch(n);
            }
            // Prepare input
            Dtype* current_top = output_at_batch(n);
            if (num_regions_ > 1) {
                split_patches_in = split_patches_in_t.data();
                split_patches_out = split_patches_out_t.data();
                split_patches_gpu<Dtype, false>(N_, K_,
                                                width_out_, height_out_, channels_out_,
                                                offsets_w_, offsets_h_, offsets_c_,
                                                shared_offsets_region_w_, shared_offsets_region_h_, shared_offsets_region_c_,
                                                col_buff, split_patches_in_t.data(), use_unshared_regions_);
            } else {
                split_patches_in = col_buff;
                split_patches_out = current_top;
            }

            // Calculate
            mex_forward_gpu<Dtype>(M_, region_size_, K_, softmax_mode_, epsilon,
                                   offsets, split_patches_in, split_patches_out, num_regions_);
            // Copy to output if needed
            if (num_regions_ > 1) {
                split_patches_gpu<Dtype, true>(N_, M_,
                                               width_out_, height_out_, channels_out_,
                                               offsets_w_, offsets_h_, offsets_c_,
                                               shared_offsets_region_w_, shared_offsets_region_h_, shared_offsets_region_c_,
                                               current_top, split_patches_out, use_unshared_regions_);
            }
        }
    }
};

REGISTER_KERNEL_BUILDER(
        Name("Mex")
                .Device(DEVICE_GPU)
                .TypeConstraint<float>("T"),
        MEXKernelGPU<float>);
REGISTER_KERNEL_BUILDER(
        Name("Mex")
                .Device(DEVICE_GPU)
                .TypeConstraint<double>("T"),
        MEXKernelGPU<double>);
